#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <complex>
#include <memory>
#include <cstdlib>
#include <cmath>
#include <pgm.h>
#include <ctime>
#include <omp.h>

#include <fstream>
#include <cpu_info.hpp>
#include <hip/hip_runtime.h>


#define NITER 1000
#define THRESHOLD 1000

#define MAXVAL 65535




int powi (int base, unsigned int exp)
{
    int res = 1;
    while (exp) {
        if (exp & 1)
            res *= base;
        exp >>= 1;
        base *= base;
    }
    return res;
}

__device__
void mset_calc(unsigned short &k, double i, double j, double scale, double ofx, double ofy){
  /**
   * Device code used to perform the actual calculation of each pixel
   * */
    double Re, Im;
    Re = i*scale + ofx;
    Im = j*scale + ofy;
    double zRe{0.}, zIm{0.}; 
    double cRe{Re},cIm{Im}, z_1Re, z_1Im, temp_zRe, temp_zIm;

    double t;
    z_1Re = zRe;
    z_1Im = zIm;


    zRe = cRe;
    zIm = cIm;
    k = 1;
    while(k < NITER){
        //
        temp_zRe = zRe;
        temp_zIm = zIm;

        t = zRe*zRe - zIm*zIm + z_1Re + cRe;
        zIm = 2*zRe*zIm + z_1Im + cIm;

        zRe = t;
        z_1Re = temp_zRe;
        z_1Im = temp_zIm;
        if(zRe*zRe + zIm*zIm > THRESHOLD){
            break;
        }
        ++k;
        
  }


}

void HOST_mset_calc(unsigned short &k, double i, double j, double scale, double ofx, double ofy){
    double Re, Im;
    Re = (i + ofx)*scale;
    Im = (j + ofy)*scale;
    double zRe{0.}, zIm{0.}; 
    double cRe{Re},cIm{Im}, z_1Re, z_1Im, temp_zRe, temp_zIm;

    double t;
    z_1Re = zRe;
    z_1Im = zIm;


    zRe = cRe;
    zIm = cIm;
    k = 1;
    while(k < NITER){
        //
        temp_zRe = zRe;
        temp_zIm = zIm;

        t = zRe*zRe - zIm*zIm + z_1Re + cRe;
        zIm = 2*zRe*zIm + z_1Im + cIm;

        zRe = t;
        z_1Re = temp_zRe;
        z_1Im = temp_zIm;
        if(zRe*zRe + zIm*zIm > THRESHOLD){
            break;
        }
        ++k;
        
  }


}


__global__
void CudaMandelbrot(size_t width, size_t height,unsigned short* image, double scale, double cx, double cy)
{
  /**
   * Kernel used to act like a bridge between CUDA and CPU code
   * The strange thing about cuda is that, every "action" on the loop
   * is performed by a single CUDA_thread, so each CUDA_thread executes a single
   * simple task. The GPU seems to be more efficient when under full load
  */
  size_t i{blockIdx.x*blockDim.x + threadIdx.x};
  size_t j{blockIdx.y*blockDim.y + threadIdx.y};

  if(i < width && j < height){
    double ii{ (double)(i)/height - 0.5};
    double jj{ (double)(j)/width - 0.5};
    unsigned short k{0};
    mset_calc(k,ii,jj,scale,cx,cy);
    image[i*width + j] = k;
  }
}

void print_err_msg(hipError_t & err){
  if (err != hipSuccess)
    {
        fprintf(stdout, "%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char** argv)
{
  //int N{255 ? 65535 : 1};

  //default hw
  size_t height{1200};
  size_t width{1200};
  clock_t t;

  double scale{0.01}, cx{-0.7}, cy{-0.5};
  if(argc > 4){
    //argv for position of the fractal
    scale = atof(argv[2]);
    cx = atof(argv[3]);
    cy = atof(argv[4]);
  }

  if(argc > 5){
    //argv for height and width
    height=atoll(argv[5]);
    width=atoll(argv[6]);
  }

  //allocate buffer on host
  unsigned short* myimg = new unsigned short[height*width];
  std::cout << "Img dimensions: h = " << height << "  w = " << width << "\n";
  std::cout << "Paramenters: scale = "<< scale << " cx = " << cx << " cy = " << cy << std::endl;

  /**everything follows is compiled only in "profiling mode, allocates a new buffer for 
   * storing the image calculated on the cpu, not needed if you do not use that
    */
  #ifdef PROF
    unsigned short* myimg_CPU = new unsigned short[height*width];
    std::cout << "Running mandelbrot (like) set calculation with time profiling: CPU vs GPU" << std::endl;
   // std::cout << "using threads on CPU " << get_cpu_info() << std::endl;
    
    t = clock();
    #pragma omp parallel
    {
      #pragma omp single
      {
        std::cout << "Using " << omp_get_num_threads() << " threads on CPU " << get_cpu_info() << std::endl;
      }
      #pragma omp for
      for(int i=0; i< height; ++i){
        for(int j=0; j< width; ++j){
          double ii{ (double)(i)/height - 0.5};
          double jj{ (double)(j)/width - 0.5};
          unsigned short k{0};
          HOST_mset_calc(k,ii,jj,scale,cx,cy);
          myimg_CPU[i*width + j] = k;
        }
      }
    }
    std::cout << "*** Elapsed calculation time CPU: " << (double)(clock() - t)/CLOCKS_PER_SEC << std::endl;
    std::cout  << "Printing a number only to trick gcc to compile" << " " << myimg_CPU[1] << std::endl;

  #endif


  
  hipError_t err = hipSuccess;

  //generating number of threads to spawn on GPU
  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);
  
  //allocate array on GPU and check for errors
  unsigned short* gpuImg;
  err = hipMalloc((void**)&gpuImg,width*height*2);

  print_err_msg(err);

  //let GPU calculate mandelbrot set and check for errors
  
  t = clock();

  CudaMandelbrot<<<numBlocks, threadsPerBlock>>>(width, height, gpuImg, scale, cx, cy);

  
  
  err = hipGetLastError();
  std::cout << "*** Elapsed calculation time GPU: " << (double)(clock() - t)/CLOCKS_PER_SEC << std::endl;

  

  print_err_msg(err);

  //copy back pgm image
  t = clock();
  err = hipMemcpy(myimg,gpuImg,height*width*2, hipMemcpyDeviceToHost);
  std::cout << "*** Time to copy the result from GPU: " << (double)(clock() - t)/CLOCKS_PER_SEC << std::endl;
  print_err_msg(err);

  //check for img being equal
  /*
  #ifdef PROF
  for(int i = 0; i< height*width;++i){
    if(myimg[i] != myimg_CPU[i]){
      std::cout << "err" << myimg[i] - myimg_CPU[i] << std::endl;
    }
  }
  #endif
  */
  //wirte pgm image on file
  pgm_img<unsigned short> test_img{height,width,myimg};
  
  std::cout << "writing image on file ";

  if(argc > 1){
    write_pgm(test_img, argv[1]);
    std::cout << argv[1] << std::endl;
    
  }
  else{
    write_pgm(test_img, "img.ppm");
    std::cout << "img.pgm" << std::endl;
  }

  hipFree(gpuImg);
  std::cout << "End" << std::endl;
  
  return 0;
}